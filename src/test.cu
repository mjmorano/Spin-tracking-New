#include "hip/hip_runtime.h"
#include "../include/test.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void cuda_kernel(double *A, double *B, double *C, int arraySize){
    // Get thread ID.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread is within array bounds.
    if ( tid < arraySize ) {
        // Add a and b.
        C[tid] = A[tid] + B[tid];
    }
}

void kernel(double *A, double *B, double *C, int arraySize) {

    // Initialize device pointers.
    double *d_A, *d_B, *d_C;

    // Allocate device memory.
    hipMalloc((void**) &d_A, arraySize * sizeof(double));
    hipMalloc((void**) &d_B, arraySize * sizeof(double));
    hipMalloc((void**) &d_C, arraySize * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.
    cuda_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, arraySize);

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
}
